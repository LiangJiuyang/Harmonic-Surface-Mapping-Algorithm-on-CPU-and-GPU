#include "hip/hip_runtime.h"
#include<stdio.h>
#include"CalculateFarField.h"
#include<hip/hip_runtime.h>
#include "hipblas.h"
#include <pthread.h>//head file needed for thread operation
using namespace std;

//MKL macro definition
#ifndef lapack_int
_#define lapack_int MKL_INT
#endif
#ifndef lapack_logical
_#define lapack_logical lapack_int
#endif

extern void line_fun(double **a,double *b,double *ppp);
int BlockNum;//Number of blocks
int ThreadNum;//Number of threads

/*
    Set number of GPU. Note that this code is based on two GPU whose cudaStatus number are 0 and 2. 
    User need to find your own GPU's cudaStatus number.
    For example, one have a GPU which cudaStatus 0, all of the code "cudaStatus = hipSetDevice(2)" need to be changed to "cudaStatus = hipSetDevice(0)".  
*/
hipError_t cudaStatus;
 
//bodyBody interaction qj/|ri-rj|
__device__ double bodyBodyInteraction(double4 bi, double4 bj, double ai)
{
	double3 r;

	//r_ij 
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;

	double distSqr = r.x*r.x + r.y*r.y + r.z*r.z;

	double invDistCube = (bj.w) / sqrt(distSqr);

	if (r.x == 0 && r.y == 0 && r.z == 0)
	  {
		  invDistCube = 0.0;
	  }
	ai = ai + invDistCube;
	return ai;
}

//tile calculation(sum of body-body interactions in a tile)
__device__ double tile_calculation(double4 myPosition, double accel)
{
	int i;
	extern __shared__ double4 shPosition[];
	for (i = 0; i<blockDim.x; i++)
	{
		accel = bodyBodyInteraction(myPosition, shPosition[i], accel);
	}
	return accel;
}

//sum of body-body interactions in all of the tiles
__global__ void calculate_forces(double4 *devX, double4 *devY, double *devA, int *Number)
{
	extern __shared__ double4 shPosition[];
	double4 myPosition;
	int i, tile;
	double acc = 0.0;
	int gtid = blockIdx.x*blockDim.x + threadIdx.x;

	if (gtid<Number[0])
	{
		myPosition = devX[gtid];
	}

	for (i = 0, tile = 0; i<Number[1]; i += blockDim.x, tile++)
	{
		int idx = tile*blockDim.x + threadIdx.x;

		if (idx<Number[1])
		{
			shPosition[threadIdx.x] = devY[idx];
		}
		else if (idx >= Number[1])
		{
			shPosition[threadIdx.x].x = 0.00;
			shPosition[threadIdx.x].y = 0.00;
			shPosition[threadIdx.x].z = 0.00;
			shPosition[threadIdx.x].w = 0.00;
		}

		__syncthreads();

		if (gtid<Number[0])
		{
			acc = tile_calculation(myPosition, acc);
		}

		__syncthreads();
	}

	devA[gtid] = acc;

}

//Energy calculation(these three components are many same as body-body interaction) qiqj/|ri-rj|
__device__ double bodyBodyInteraction1(double4 bi, double4 bj, double ai)
{
	double3 r;

	//r_ij 
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;

	double distSqr = r.x*r.x + r.y*r.y + r.z*r.z;
	double invDistCube = (bi.w+0.00)*(bj.w + 0.00) / sqrt(distSqr);
	if (r.x == 0 && r.y == 0 && r.z == 0)
	  {
		  invDistCube = 0.0;
	  }
	ai = ai + invDistCube;
	return ai;
}

__device__ double tile_calculation1(double4 myPosition, double accel)
{
	int i;
	extern __shared__ double4 shPosition[];
	for (i = 0; i<blockDim.x; i++)
	  {
	   	accel = bodyBodyInteraction1(myPosition, shPosition[i], accel);
  	}
	return accel;
}

__global__ void calculate_forces1(double4 *devX, double4 *devY, double *devA, int *Number)
{
	extern __shared__ double4 shPosition[];
	double4 myPosition;
	int i, tile;
	double acc = 0.0;
	int gtid = blockIdx.x*blockDim.x + threadIdx.x;
	if (gtid<Number[0])
	{
		myPosition = devX[gtid];
	}
	else if(gtid>=Number[0])
	{
	        myPosition.x = 1.00;
			myPosition.y = 1.00;
			myPosition.z = 1.00;
			myPosition.w = 0.00;
	}

	for (i = 0, tile = 0; i<Number[1]; i += blockDim.x, tile++)
	{
		int idx = tile*blockDim.x + threadIdx.x;

		if (idx<Number[1])
		{
			shPosition[threadIdx.x] = devY[idx];
		}
		else if (idx >= Number[1])
		{
			shPosition[threadIdx.x].x = 0.00;
			shPosition[threadIdx.x].y = 0.00;
			shPosition[threadIdx.x].z = 0.00;
			shPosition[threadIdx.x].w = 0.00;
		}

		__syncthreads();
	}

	devA[gtid] = acc;	
}

//Calculate factorial on GPU 
__device__ double facGpu(double ttt)
{
    double s;
    if (abs(ttt - 1)<0.001 || abs(ttt)<0.001)
		s = 1.0;
	else
	{ 
		s = ttt*facGpu(ttt - 1) + 0.00;
	}
	return s;
}

//Calculate multipole expansion on GPU
__global__ void Calculate_MultipoleExpansion(double *ExpansionMatrix,double4 *PointPosition,int *QuizPointNumber,int *ExpansionNumber)
{
           double4 myPosition;
           int gtid = blockIdx.x*blockDim.x + threadIdx.x;
           myPosition=PointPosition[gtid];
           double x,y,z;
           x=myPosition.x;
           y=myPosition.y;
           z=myPosition.z;
           if(gtid<QuizPointNumber[0]) 
       {             
	ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+0] = 1.0;
	ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+1] = y / 2;
	ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+2] =(-z);
	ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+3] =(-x / 2);
	int t = 4;
	int m, n,i;
	for (i = 2; i < ExpansionNumber[0]; i++) 
	{
		while (t<(i+1)*(i+1))
		{
			m = t - i - i*i;
			n = i; 
			if (m==-n)
			{
				ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+t] =(y*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n - 1] - x*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n - 2 * n + 1]) / (2 * n + 0.00);
			}
			else if (m==n)
			{
				ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+t] = (-(x*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n - 1] + y*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n - 2 * n + 1]) / (2 * n + 0.00));
			}
			else if (n - abs(m) == 1)
			{
				ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+t] = (-z)*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n-n+m];
			}      
			else if ((n-abs(m))>1)    
			{
				ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+t] = (-((2 * n - 1.0)*z*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n-n+m] + (x*x + y*y + z*z)*ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+n*n-3*n+m+2]) / ((n - abs(m)+0.0)*(n + abs(m)+0.0)));
			}
			t++;
		}
	}

	t = 0;
	for (i = 0; i < ExpansionNumber[0]; i++)
	{
		while (t < (i + 1)*(i + 1))
		{
			m = t - i - i*i;
			n = i;
			ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+t] = ExpansionMatrix[gtid*ExpansionNumber[0]*ExpansionNumber[0]+t] * sqrt((facGpu(n - m+0.0))*facGpu(n + m+0.0));
			t++;
		}
	}
       }
}

//Output device information
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
    //used to count the device numbers
    int count;

    // get the cuda device count
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    // find the device >= 1.X

    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 0) {
                printDeviceProp(prop);
                break;
            }
        }
    }

    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    // set cuda device
    hipSetDevice(i);
    return true;
}



int main()
{
  //Initial GPU
  if (!InitCUDA()) 
     {
       return 0;
     }
  cudaStatus = hipSetDevice(2);
  int *kkkkl;
	hipMalloc((void**)&kkkkl, sizeof(int));
  hipFree(kkkkl); 
       
	hipError_t cudaStatus;  
  int num;   
  cudaStatus = hipGetDeviceCount(&num);
	cout<<"Num = "<<num<<endl; 
	   
  //Set Fibonacci Integral's parameter   
  Fp = 610.0;
  F = 987.0;
	Np = F * 2.0 + 2.0;
	deltaz = 2.0 / F;
	
  //HSMA Parameters' set
  p=25;//number of multiple expansion's truncate
  Rb=1.5*sqrt(3);//Rb is Rs in the related paper     
	QuizNumber=2*p*p;//number of test point on the surface
  const int N=CenterBoxSourceNumber;//number of particles in center box
		
  double duration;
  clock_t start, finish;//clock
                  
  
		Particle FieldCharge[N];
		SetFieldCharge(FieldCharge);//set charge of field source
		QuizPointSum PointSum;// quiz points set (has been constructed in the constructor)


		cout << "Algorithm include FMM : " << endl;
		cout << "N = " << N << endl;
		cout << "p= " << p << endl;
		cout << "Rb = " << (Rb+0.00)/sqrt(3)<<"*sqrt(3)" << endl;
		cout << "QuizPoints=" << QuizNumber << endl;
		cout << "Np = " << Np << endl;
		cout << "DeltaR=" << DeltaR << endl; 
		 
		clock_t startFULL, finishFULL;//clock
		startFULL = clock();
    start=clock();
		
    QuizPointSum QuizSum;//Corresponding quiz points which is moved into center box Omega0 from PointSum that on the circle S0
		QuizSum = FindCorrespondingPoints(PointSum);

		//Find all of the period field charges
		int xx1 = int((Rb - L1 / 2) / L1) + 1;//px=L1,2*L1,...,xx1*L1
		int xx2 = int((Rb - L2 / 2) / L2) + 1;//py=L2,2*L2,...,xx2*L2
		int xx3 = int((Rb - L3 / 2) / L3) + 1;//pz=L3,2*L3,...,xx3*L3
		int BBoxSum = (2 * xx1 + 1)*(2 * xx2 + 1)*(2 * xx3 + 1);//number of box which can wrap sphere Sb
		Particle *PP = new Particle[BBoxSum*N];
		FindAllFieldPointSum(Rb, L1, L2, L3, FieldCharge, PP);//Find all of the field charges. set PP
 
                   
		int qq = 0;
		Particle *PPP = new Particle[BBoxSum*N];
		qq = AdjustAllFieldPointSum(PP, BBoxSum*N, PPP);//Find field charge in Rs
		Particle *YY = new Particle[qq];
		for (int i = 0; i < qq; i++)
		{
			YY[i] = PPP[i];//Field charges in Rs are set in YY
		}

		double *f=new double[QuizNumber];//store right-hand term
  
    finish = clock();
		printf("Time for CPU to initial set is:");
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration);

               

/*      begin    cuda       */
    start=clock();
		//Set GPU's Number 
    // Choose which GPU to run on, change this on a multi-GPU system.  
    cudaStatus = hipSetDevice(1);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");    
        } 

		/*   The First GPU   */
		//CPUset
		double(*QuizPointCopy)[4] = (double(*)[4])malloc(QuizNumber * 2 * sizeof(double[4]));//change structure for gpu
		for (int i = 0; i < QuizNumber; i++)
		{
			QuizPointCopy[i][0] = QuizSum.x[i];
			QuizPointCopy[i][1] = QuizSum.y[i];
			QuizPointCopy[i][2] = QuizSum.z[i];
			QuizPointCopy[i][3] = 0.00;
			QuizPointCopy[i + QuizNumber][0] = PointSum.x[i];
			QuizPointCopy[i + QuizNumber][1] = PointSum.y[i];
			QuizPointCopy[i + QuizNumber][2] = PointSum.z[i];
			QuizPointCopy[i + QuizNumber][3] = 0.00;
		}
		//GPUset
		double4 *PositionForQuizPoint;
		double4 *Source,*Source1;//We will use Source twice. So that it do not need to free quickly.
 
		hipMalloc((void**)&PositionForQuizPoint, sizeof(double4)*QuizNumber * 2);
		//hipMalloc((void**)&PositionForPoint, sizeof(double4)*QuizNumber);
		hipMalloc((void**)&Source, sizeof(double4)*qq);
		hipMalloc((void**)&Source1, sizeof(double4)*qq/2);
		hipMemcpy(PositionForQuizPoint, QuizPointCopy, sizeof(double4)*QuizNumber * 2, hipMemcpyHostToDevice);
		hipMemcpy(Source, YY, sizeof(double4)*qq, hipMemcpyHostToDevice);
		hipMemcpy(Source1, YY, sizeof(double4)*qq/2, hipMemcpyHostToDevice);


		int number[2];
		int *Number;
		number[0] = QuizNumber * 2;
		number[1] = qq/2;
		hipMalloc((void**)&Number, sizeof(int) * 2);
		hipMemcpy(Number, number, sizeof(int) * 2, hipMemcpyHostToDevice);

		double *GPUQuizPointPotential;
		hipMalloc((void**)&GPUQuizPointPotential, sizeof(double)*QuizNumber * 2);

		BlockNum = QuizNumber*2/128+1;
		ThreadNum = 128;

		//Calculate
		calculate_forces << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (PositionForQuizPoint, Source1, GPUQuizPointPotential, Number);


		//Copy from GPU to CPU
		double GPUQUIZPOINTPOTENTIAL[QuizNumber * 2];

    //Initial the second GPU and substract it from the total time
    clock_t cutstart,cutfinish;
    cutstart=clock();
    cudaStatus = hipSetDevice(0); 
  	int *kkkl;
	  hipMalloc((void**)&kkkl, sizeof(int));
	  hipFree(kkkl);
    cutfinish=clock();
    double cut1;
    cut1=(double)(cutfinish - cutstart) / CLOCKS_PER_SEC;
    cout<<"cut1 = "<<cut1<<endl;

		/*   The Second GPU   */
		double4 *PositionForQuizPoint2;
		double4 *Source2;//We will use Source twice. So that it do not need to free quickly.
		hipMalloc((void**)&PositionForQuizPoint2, sizeof(double4)*QuizNumber * 2);
		hipMalloc((void**)&Source2, sizeof(double4)*(qq-qq/2));
		hipMemcpy(PositionForQuizPoint2, QuizPointCopy, sizeof(double4)*QuizNumber * 2, hipMemcpyHostToDevice);
		hipMemcpy(Source2, &YY[qq/2], sizeof(double4)*(qq-qq/2), hipMemcpyHostToDevice);

                  
		int *Number2;
		number[0] = QuizNumber * 2;
		number[1] = qq-qq/2;
		hipMalloc((void**)&Number2, sizeof(int) * 2);
		hipMemcpy(Number2, number, sizeof(int) * 2, hipMemcpyHostToDevice);
		double *GPUQuizPointPotential2;
		hipMalloc((void**)&GPUQuizPointPotential2, sizeof(double)*QuizNumber * 2);
		BlockNum = QuizNumber*2/128+1;
		ThreadNum = 128;
		//Calculate
		calculate_forces << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (PositionForQuizPoint2, Source2, GPUQuizPointPotential2, Number2);

    //Copy from GPU to CPU
		double GPUQUIZPOINTPOTENTIAL2[QuizNumber * 2];
		hipMemcpy(&GPUQUIZPOINTPOTENTIAL, GPUQuizPointPotential, sizeof(double)*QuizNumber * 2, hipMemcpyDeviceToHost);
		hipMemcpy(&GPUQUIZPOINTPOTENTIAL2, GPUQuizPointPotential2, sizeof(double)*QuizNumber * 2, hipMemcpyDeviceToHost);


		
/*                    Prepare for next use                                   */
    double4 *SourceSecond;//We will use Source twice. So that it do not need to free quickly.
    hipMalloc((void**)&SourceSecond, sizeof(double4)*qq); 
    hipMemcpy(SourceSecond, YY, sizeof(double4)*qq, hipMemcpyHostToDevice);
    double4 *PositionFieldSecond;//We will use it twice
		hipMalloc((void**)&PositionFieldSecond, sizeof(double4)*N/2);
		hipMemcpy(PositionFieldSecond, &FieldCharge[N/2], sizeof(double4)*N/2, hipMemcpyHostToDevice);
    int *NumberSecond;
		hipMalloc((void**)&NumberSecond, sizeof(int)*2);
		number[0] = qq;
		number[1] = N/2;
		hipMemcpy(NumberSecond, number, sizeof(int) * 2, hipMemcpyHostToDevice);
/*                    End prepare                                      */


		//Free
		hipFree(PositionForQuizPoint); hipFree(GPUQuizPointPotential);
/*       end      cuda       */

  
		for (int i = 0; i < QuizNumber; i++)
		{
			f[i] = GPUQUIZPOINTPOTENTIAL[i] - GPUQUIZPOINTPOTENTIAL[i + QuizNumber]+GPUQUIZPOINTPOTENTIAL2[i] - GPUQUIZPOINTPOTENTIAL2[i + QuizNumber];
		}


    finish = clock();
		printf("Time for GPU to calculate the right part is:");
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration-cut1);


    start=clock();
    cudaStatus = hipSetDevice(1);
    double *QuizSumExpansionMatrix;
    hipMalloc((void **)&QuizSumExpansionMatrix,sizeof(double)*QuizNumber*p*p);
    double4 *QuizSumPosition;              
    hipMalloc((void **)&QuizSumPosition,sizeof(double4)*QuizNumber);
    hipMemcpy(QuizSumPosition, QuizPointCopy, sizeof(double4) * QuizNumber, hipMemcpyHostToDevice);
    int  *QuizNumberCuda1;
    int  *ExpansionNumberCuda1; 
    hipMalloc((void **)&QuizNumberCuda1,sizeof(int));                                
    hipMalloc((void **)&ExpansionNumberCuda1,sizeof(int));
    int *UM=new int[2];
    UM[0]=QuizNumber;
    UM[1]=p;
    hipMemcpy(QuizNumberCuda1,&UM[0],sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(ExpansionNumberCuda1,&UM[1],sizeof(int),hipMemcpyHostToDevice);     
    BlockNum=QuizNumber/128+1;
    ThreadNum=128;
    Calculate_MultipoleExpansion<<<BlockNum,ThreadNum,0>>>(QuizSumExpansionMatrix,QuizSumPosition,QuizNumberCuda1,ExpansionNumberCuda1);

    cudaStatus = hipSetDevice(0);
    double *PointSumExpansionMatrix;
    hipMalloc((void **)&PointSumExpansionMatrix,sizeof(double)*QuizNumber*p*p);
    double4 *PointSumPosition;
    hipMalloc((void **)&PointSumPosition,sizeof(double4)*QuizNumber); 
    hipMemcpy(PointSumPosition, &QuizPointCopy[QuizNumber], sizeof(double4) * QuizNumber, hipMemcpyHostToDevice);                                                  
    int *QuizNumberCuda2,*ExpansionNumberCuda2;
    hipMalloc((void **)&QuizNumberCuda2,sizeof(int)); 
    hipMalloc((void **)&ExpansionNumberCuda2,sizeof(int));
    hipMemcpy(QuizNumberCuda2,&UM[0],sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(ExpansionNumberCuda2,&UM[1],sizeof(int),hipMemcpyHostToDevice);                   
    Calculate_MultipoleExpansion<<<BlockNum,ThreadNum,0>>>(PointSumExpansionMatrix,PointSumPosition,QuizNumberCuda2,ExpansionNumberCuda2);
                  
                    
    //Calculate Projection in cpu
		ProjectPointSum D;//D is points on R'
		ProjectPointSum DD;//DD is corresponding points on R''
    ProjectPointSum DDD;//DD is corresponding points on R'''
    double4 *D4=new double4[Np];
		for (int i = 0; i < Np; i++)
		{
			DD.x[i] = D.x[i] * (Rb + DeltaR/2.0 + 0.0) / (Rb + 0.0);//Rb+0.00 is needed. Please do not use Rb only
			DD.y[i] = D.y[i] * (Rb + DeltaR/2.0 + 0.0) / (Rb + 0.0);
			DD.z[i] = D.z[i] * (Rb + DeltaR/2.0 + 0.0) / (Rb + 0.0);
      DDD.x[i] = D.x[i] * (Rb - DeltaR/2.0 + 0.0) / (Rb + 0.0);//Rb+0.00 is needed. Please do not use Rb only
			DDD.y[i] = D.y[i] * (Rb - DeltaR/2.0 + 0.0) / (Rb + 0.0);
			DDD.z[i] = D.z[i] * (Rb - DeltaR/2.0 + 0.0) / (Rb + 0.0);
      D4[i].x=D.x[i];
      D4[i].y=D.y[i];
      D4[i].z=D.z[i];
      D4[i].w=0.00;
		} 
                  
		//Prepare for calculate far field potential
		double *Sigma_r = new double[Np];//Sigma_r is for sigma(r')
		double *Sigma_rr = new double[Np];//Sigma_rr is for sigma(r'')
		double kkk;
  //End Calculate in cpu    


    double A[QuizNumber][p*p];
    double AA[QuizNumber][p*p];
    cudaStatus = hipSetDevice(1);
    hipMemcpy(&A, QuizSumExpansionMatrix, sizeof(double)*QuizNumber*p*p, hipMemcpyDeviceToHost);                       
    cudaStatus = hipSetDevice(0);
    hipMemcpy(&AA, PointSumExpansionMatrix, sizeof(double)*QuizNumber*p*p, hipMemcpyDeviceToHost);       
    hipFree(QuizSumExpansionMatrix);
    hipFree(PointSumExpansionMatrix);
    hipFree(QuizSumPosition);hipFree(QuizNumberCuda1);hipFree(ExpansionNumberCuda1);hipFree(PointSumPosition);hipFree(QuizNumberCuda2);hipFree(ExpansionNumberCuda2);
                                  
    for(int i=0;i<QuizNumber;i++)
    for(int j=0;j<p*p;j++)
      {
        A[i][j]=AA[i][j]-A[i][j];
      }
  
    finish = clock();
		printf("Time for CPU to calculate multi-pole expansion is:");
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration);            

    start=clock();   
                   
    cudaStatus = hipSetDevice(1);
    double *DSumExpansionMatrix;
    hipMalloc((void **)&DSumExpansionMatrix,sizeof(double)*(Np/2)*p*p);
    double4 *DSumPosition;              
    hipMalloc((void **)&DSumPosition,sizeof(double4)*(Np/2));
    hipMemcpy(DSumPosition, D4, sizeof(double4) * (Np/2), hipMemcpyHostToDevice);
    int  *DNumberCuda1;
    int  *DExpansionNumberCuda1; 
    hipMalloc((void **)&DNumberCuda1,sizeof(int));                                
    hipMalloc((void **)&DExpansionNumberCuda1,sizeof(int));
    UM[0]=Np/2;
    UM[1]=p;
    hipMemcpy(DNumberCuda1,&UM[0],sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(DExpansionNumberCuda1,&UM[1],sizeof(int),hipMemcpyHostToDevice);     
    BlockNum=(Np/2)/64+1;
    ThreadNum=64;
    Calculate_MultipoleExpansion<<<BlockNum,ThreadNum,0>>>(DSumExpansionMatrix,DSumPosition,DNumberCuda1,DExpansionNumberCuda1);
                     
    cudaStatus = hipSetDevice(0); 
    BlockNum=(Np-Np/2)/64+1;
    ThreadNum=64;
    double *D2SumExpansionMatrix;
    hipMalloc((void **)&D2SumExpansionMatrix,sizeof(double)*(Np-Np/2)*p*p);
    double4 *D2SumPosition;   
    hipMalloc((void **)&D2SumPosition,sizeof(double4)*(Np-Np/2));
    hipMemcpy(D2SumPosition, &D4[Np/2], sizeof(double4) * (Np-Np/2), hipMemcpyHostToDevice);                                                    
    int *D2NumberCuda2,*D2ExpansionNumberCuda2;
    hipMalloc((void **)&D2NumberCuda2,sizeof(int)); 
    hipMalloc((void **)&D2ExpansionNumberCuda2,sizeof(int));   
    UM[0]=Np-Np/2;
    UM[1]=p;
    hipMemcpy(D2NumberCuda2,&UM[0],sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(D2ExpansionNumberCuda2,&UM[1],sizeof(int),hipMemcpyHostToDevice);                   
    Calculate_MultipoleExpansion<<<BlockNum,ThreadNum,0>>>(D2SumExpansionMatrix,D2SumPosition,D2NumberCuda2,D2ExpansionNumberCuda2);

    double AB[Np][p*p];
    cudaStatus = hipSetDevice(1);
    hipMemcpy(&AB, DSumExpansionMatrix, sizeof(double)*(Np/2)*p*p, hipMemcpyDeviceToHost);            
    cudaStatus = hipSetDevice(0);
    hipMemcpy(&AB[Np/2][0], D2SumExpansionMatrix, sizeof(double)*(Np-Np/2)*p*p, hipMemcpyDeviceToHost);        
    hipFree(DSumExpansionMatrix);   hipFree(DSumPosition);  hipFree(DNumberCuda1);  hipFree(DExpansionNumberCuda1); 
    hipFree(D2SumExpansionMatrix);   hipFree(D2SumPosition);  hipFree(D2NumberCuda2);  hipFree(D2ExpansionNumberCuda2);                  

    finish = clock();
		printf("Time for CPU to calculate projection is:");
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration);
  
    clock_t start1, finish1;//clock
		start1 = clock();	
                  
		//Calculate Near Field Potential
/*                begin         GPU                      */
		cudaStatus = hipSetDevice(1);  
    /*      The First GPU     */
		//CPUset
		//We have copied YY to GPU.(Source) And we can copy FieldCharge to GPU directly.
		//GPUset
                                 
		double4 *PositionField;//We will use it twice
		hipMalloc((void**)&PositionField, sizeof(double4)*N/2);
		hipMemcpy(PositionField, FieldCharge, sizeof(double4)*N/2, hipMemcpyHostToDevice);
        
		number[0] = qq;
		number[1] = N/2;
		hipMemcpy(Number, number, sizeof(int) * 2, hipMemcpyHostToDevice);

		double *GPUFieldPotential;//We will use it twice
		hipMalloc((void**)&GPUFieldPotential, sizeof(double)*qq);

		BlockNum = qq / 256 + 1;
		ThreadNum = 256;
  
		//Calculate
		calculate_forces1 << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (Source,PositionField,GPUFieldPotential, Number);

		 //Copy from GPU to CPU
		double GPUFIELDPOTENTIAL[qq];//If N=216000, we need to set stack 81920Kbts.
		clock_t time_use1[BlockNum*2];
                  	
		/*      The second gpu      */
    cudaStatus = hipSetDevice(0); 
		//CPUset
		//We have copied YY to GPU.(Source) And we can copy FieldCharge to GPU directly.
		//GPUset

		double *GPUFieldPotentialSecond;//We will use it twice
		hipMalloc((void**)&GPUFieldPotentialSecond, sizeof(double)*qq);
                  
		BlockNum = qq / 256 + 1;
		ThreadNum = 256;

		//Calculate
		calculate_forces1 << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (SourceSecond,PositionFieldSecond,GPUFieldPotentialSecond, NumberSecond);
 
		 //Copy from GPU to CPU
		double GPUFIELDPOTENTIALSECOND[qq];//If N=216000, we need to set stack 81920Kbts.
		
/*            end    GPU                */
                   
/*                       Begin calculate for multi-pole expansion                          */
                 
    start=clock();
		//A is the matrix of Quiz - Point
    //Set augmented matrix(used for LS least Squares)
		double(*B)[p*p - 1] = (double(*)[p*p - 1])malloc(QuizNumber * sizeof(double[p*p - 1]));
		for (int i = 0; i < QuizNumber; i++)
		{
			for (int j = 0; j < p*p - 1; j++)
			{
				if (j < p*p - 1)
				{
					B[i][j] = A[i][j + 1];
				}
			}
		}

            
/*                        End calculate for multi-pole expansion                            */
    start = clock();
                   
		double *C=new double[p*p];//Storage Solution  

/*                           Solve      Least      Square                             */
    //Using MKL for accerate 
                 
    double *MatrixAT,*MatrixATA;
    int rowAT,columnATA,columnAT; 
    double alpha,beta;
    rowAT=p*p-1;columnAT=QuizNumber;columnATA=p*p-1;
    alpha=1.0;beta=0.00;
    MatrixAT = (double *)mkl_malloc( rowAT*columnAT*sizeof( double ), 64 );
    MatrixATA=(double *)mkl_malloc(rowAT*columnATA*sizeof(double),64);
    if (MatrixAT == NULL || MatrixATA == NULL) {
      printf( "\n ERROR: Can't allocate memory for matrices. Aborting... \n\n");
      mkl_free(MatrixAT);
      mkl_free(MatrixATA);
      return 1;
    }
    for(int i=0;i<rowAT;i++)
       for(int j=0;j<columnAT;j++)
           {
              MatrixAT[i*columnAT+j]=B[j][i];
            }
    for (int i = 0; i < (rowAT*columnATA); i++) {
         MatrixATA[i] = 0.0;
    }
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasTrans, rowAT, columnATA, columnAT, alpha, MatrixAT, columnAT, MatrixAT, columnAT, beta, MatrixATA, columnATA);
    int InfoHelp,*VectorHelp;
    VectorHelp=(int *)mkl_malloc(columnATA*sizeof(int),64); 
    for(int i=0;i<columnATA;i++)
        VectorHelp[i]=0;
    InfoHelp=LAPACKE_dgetrf(CblasRowMajor,columnATA,columnATA,MatrixATA,columnATA,VectorHelp);
    InfoHelp=LAPACKE_dgetri(CblasRowMajor,columnATA,MatrixATA,columnATA,VectorHelp); 
    double *BB,*ATB,*INV_ATA_ATB;
    BB = (double *)mkl_malloc( columnAT*1*sizeof( double ), 64 );
    ATB = (double *)mkl_malloc( columnATA*1*sizeof( double ), 64 );
    INV_ATA_ATB = (double *)mkl_malloc( columnATA*1*sizeof( double ), 64 );
    for(int i=0;i<columnAT;i++)
        {
           BB[i]=f[i];
        }
    for(int i=0;i<columnATA;i++)
        {
           ATB[i]=0.00;
           INV_ATA_ATB[i]=0.00;
        }
                                                               
    cblas_dgemv(CblasRowMajor, CblasNoTrans, rowAT,columnAT, alpha, MatrixAT, columnAT, BB, 1, beta, ATB, 1);
    cblas_dgemv(CblasRowMajor, CblasNoTrans,columnATA,columnATA, alpha, MatrixATA, columnATA, ATB, 1, beta, INV_ATA_ATB, 1);
                  
		/*                      End        Least      Square                         */
		for (int i = 1; i < p*p; i++)
			C[i] = INV_ATA_ATB[i-1];
      C[0] = 0;
                                                      
		finish = clock();
		printf("Time to solve the LS Least Squares is:");
		duration = (double)(finish - start) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration);

    //Calculate Parameter for Harmonic Mapping
		for (int i = 0; i < Np; i++)
		{
			Sigma_r[i] = 0.00;
			Sigma_rr[i] = 0.00;
			for (int j = 0; j < p*p; j++)
			{
				kkk = (floor(sqrt(j)) + 0.00);
				Sigma_rr[i] = Sigma_rr[i] + C[j] * AB[i][j] / (DeltaR + 0.00);
				Sigma_r[i] = Sigma_r[i] + C[j] * AB[i][j] * (kkk / (Rb + 0.0));
			}
		 }
                 	
        

		/*       begin CUDA            */
    cudaStatus = hipSetDevice(1);  
    double4 *PositionField1;//We will use it twice
		hipMalloc((void**)&PositionField1, sizeof(double4)*N);
		hipMemcpy(PositionField1, FieldCharge, sizeof(double4)*N, hipMemcpyHostToDevice);

		int *Number1;
		hipMalloc((void**)&Number1, sizeof(int) * 2);

		//CPUset
		double(*DCopy)[4] = (double(*)[4])malloc(Np * sizeof(double[4]));
    double(*DDCopy)[4] = (double(*)[4])malloc(Np * sizeof(double[4]));
    double(*DDDCopy)[4] = (double(*)[4])malloc(Np * sizeof(double[4]));
		for (int i = 0; i < Np; i++)
		{ 
			DCopy[i][0] = D.x[i];
			DCopy[i][1] = D.y[i];
			DCopy[i][2] = D.z[i];
			DCopy[i][3] = ((Rb + 0.0)*(Rb + 0.0))*D.q[i] * Sigma_r[i] / (4 * pi);
			DDCopy[i][0] = DD.x[i];
			DDCopy[i][1] = DD.y[i];
			DDCopy[i][2] = DD.z[i];
			DDCopy[i][3] = ((Rb + 0.0)*(Rb + 0.0))*DD.q[i] * Sigma_rr[i] / (4 * pi);
	                  DDDCopy[i][0] = DDD.x[i];
			DDDCopy[i][1] = DDD.y[i];
			DDDCopy[i][2] = DDD.z[i];
			DDDCopy[i][3] = ((Rb + 0.0)*(Rb + 0.0))*DDD.q[i] * Sigma_rr[i] / (4 * pi);
		} 
		//GPUset
		double4 *SourceD,*SourceDD,*SourceDDD;
		hipMalloc((void**)&SourceD,sizeof(double4)*Np);
		hipMemcpy(SourceD,DCopy,sizeof(double4)*Np,hipMemcpyHostToDevice);
    hipMalloc((void**)&SourceDD,sizeof(double4)*Np);
		hipMemcpy(SourceDD,DDCopy,sizeof(double4)*Np,hipMemcpyHostToDevice);
    hipMalloc((void**)&SourceDDD,sizeof(double4)*Np);
		hipMemcpy(SourceDDD,DDDCopy,sizeof(double4)*Np,hipMemcpyHostToDevice);
		
		number[0] = N;
		number[1] = Np;
		hipMemcpy(Number1, number, sizeof(int) * 2, hipMemcpyHostToDevice);

		BlockNum = N / 128 + 1;
		ThreadNum = 128;
 
    double *GPUFieldPotentialD,*GPUFieldPotentialDD,*GPUFieldPotentialDDD;
		hipMalloc((void**)&GPUFieldPotentialD, sizeof(double)*N);
    hipMalloc((void**)&GPUFieldPotentialDD, sizeof(double)*N);
    hipMalloc((void**)&GPUFieldPotentialDDD, sizeof(double)*N);

		//Calculate
		calculate_forces1 << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (PositionField1, SourceD, GPUFieldPotentialD, Number1);
		calculate_forces1 << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (PositionField1, SourceDD, GPUFieldPotentialDD, Number1);
    calculate_forces1 << <BlockNum, ThreadNum, sizeof(double4)*ThreadNum >> > (PositionField1, SourceDDD, GPUFieldPotentialDDD, Number1);	
			                 
		//Copy from GPU to CPU
    //near field potential
    cudaStatus = hipSetDevice(0); 

    hipMemcpy(&GPUFIELDPOTENTIALSECOND, GPUFieldPotentialSecond, sizeof(double)*qq, hipMemcpyDeviceToHost);
		hipFree(SourceSecond);
		hipFree(PositionFieldSecond);
		hipFree(NumberSecond);
		hipFree(GPUFieldPotentialSecond);
    cudaStatus = hipSetDevice(1); 
		hipMemcpy(&GPUFIELDPOTENTIAL, GPUFieldPotential, sizeof(double)*qq, hipMemcpyDeviceToHost);
		hipFree(Source);
		hipFree(PositionField);
		hipFree(Number);
    hipFree(GPUFieldPotential);  
        
		finish1 = clock();
    double duration1;
		duration1 = (double)(finish1 - start1+0.00) / (CLOCKS_PER_SEC+0.00);
		printf("Time for GPU to calculate the near field potential is:");
		printf("%f seconds\n", duration1);
	                 
		double GPUFIELDPOTENTIALFARD[N],GPUFIELDPOTENTIALFARDD[N],GPUFIELDPOTENTIALFARDDD[N];
		hipMemcpy(&GPUFIELDPOTENTIALFARD, GPUFieldPotentialD, sizeof(double)*N, hipMemcpyDeviceToHost);
    hipMemcpy(&GPUFIELDPOTENTIALFARDD, GPUFieldPotentialDD, sizeof(double)*N, hipMemcpyDeviceToHost); 
    hipMemcpy(&GPUFIELDPOTENTIALFARDDD, GPUFieldPotentialDDD, sizeof(double)*N, hipMemcpyDeviceToHost); 
                   
		hipFree(SourceD);
    hipFree(SourceDD);
    hipFree(SourceDDD);

		hipFree(Number1);
		hipFree(PositionField1);

		finish1 = clock();
		printf("Time for GPU to calculate the far field potential is:");
		duration = ((double)(finish1 - start1+0.00)) / (CLOCKS_PER_SEC+0.00)-duration;
		printf("%f seconds\n", duration);  
           
		/*       end   CUDA            */
  
		start1=clock();

		double tt = 0.00;             
		for (int i = 0; i < N; i++)
		{
			
			tt = tt +(GPUFIELDPOTENTIALFARD[i]- GPUFIELDPOTENTIALFARDD[i]+GPUFIELDPOTENTIALFARDDD[i]);
		}
		for(int i=0;i<qq;i++)
		{
		  tt=tt+GPUFIELDPOTENTIAL[i]+GPUFIELDPOTENTIALSECOND[i]; 
		}
                   
                  
		cout << "Calculated by simulate is " << setprecision(16) << tt / (2.0) << endl;    //This is the result for Madelung
		cout << "The Absolute Error is" << abs(-470125.129478090000 - tt / 2.0) << endl;
		cout << "The Relative Error is" << abs(1 - (-470125.129478090000 / (tt / 2.0))) << endl;

    finish1 = clock();
		printf("Time for GPU to calculate the total sum is:");
		duration = (double)(finish1 - start1) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration);                

		finishFULL = clock();
		printf("Time for all of this prodecure is:");
		duration = (double)(finishFULL - startFULL) / CLOCKS_PER_SEC;
		printf("%f seconds\n", duration-cut1);
                                       
  	return 1;
}

